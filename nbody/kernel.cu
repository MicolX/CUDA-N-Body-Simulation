#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ 
float3 calculateAcceleration(float3 loc1, float3 loc2, float3 acc)
{
	float dx = loc2.x - loc1.x;
	float dy = loc2.y - loc1.y;
	float dz = loc2.z - loc1.z;
	float distSqrt = dx * dx + dy * dy + dz * dz + EPS2;
	float distCube = distSqrt * distSqrt * distSqrt;
	float invDist = G / sqrtf(distCube);
	acc.x += dx * invDist;
	acc.y += dy * invDist;
	acc.z += dz * invDist;
	acc.x *= SOFTEN;
	acc.y *= SOFTEN;
	acc.z *= SOFTEN;
	return acc;
}

__device__
float3 updateLocation(float3 loc, float time, float3 &acc, float3 &speed)
{
	loc.x += speed.x * time + acc.x * (time * time / 2);
	loc.y += speed.y * time + acc.y * (time * time / 2);
	loc.z += speed.z * time + acc.z * (time * time / 2);
	return loc;
}

__device__
float3 updateSpeed(float3 speed, float3 acc, float time)
{
	speed.x += acc.x * time;
	speed.y += acc.y * time;
	speed.z += acc.z * time;
	return speed;
}

__host__
__device__
int to1D(int x, int y, int z, int width, int height, int depth)
{
	return x + width * (y + depth * z);
}


__global__ 
void share_kernel(float3 *entities, float3 *accs, int length)
{
	__shared__ float3 share[TILE];
	unsigned int tx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + tx;
	float3 acc;
	float3 entity;

	if (idx < length)
	{
		entity = entities[idx];
		acc = accs[idx];
	
		for (size_t i = 0; i < length; i += TILE)
		{
			if (i + tx < length) 
			{
				share[tx] = entities[i + tx];
			}
			 
			__syncthreads();

			for (size_t j = 0; j < blockDim.x; j++)
			{
				if (i + j < length)
				{
					acc = calculateAcceleration(entity, share[j], acc);
				}
			}
			__syncthreads();
		}

		accs[idx] = acc;
	}
}

__global__ 
void naive_kernel(float3 *entities, float3 *accs, int length)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float3 entity, acc;
	if (idx < length)
	{
		entity = entities[idx];
		acc = accs[idx];


		for (size_t i = 0; i < length; i++)
		{
			acc = calculateAcceleration(entity, entities[i], acc);
		}
		accs[idx] = acc;
	}
}

__global__
void naive_op_kernel(float3 *pos, float3 *d_speed, float3 *d_accs, float time, int num)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num)
	{
		float3 entity = pos[idx];
		float3 speed = d_speed[idx];
		float3 acc = d_accs[idx];

		for (size_t i = 0; i < num; i++)
		{
			acc = calculateAcceleration(entity, pos[i], acc);
		}

		d_speed[idx] = updateSpeed(speed, acc, time);
		pos[idx] = updateLocation(entity, time, acc, speed);
		d_accs[idx] = acc;
	}
}

__global__
void share_op_kernel(float3 *pos, float3 *d_speed, float3 *d_accs, float time, int num)
{
	__shared__ float3 share_pos[TILE];
	unsigned int thx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + thx;
	float3 entity = pos[idx];
	float3 &speed = d_speed[idx];
	float3 &acc = d_accs[idx];

	if (idx < num)
	{
		for (size_t i = 0; i < num; i += TILE)
		{
			if (i + thx < num)
			{
				share_pos[thx] = pos[i + thx];
			}
			
			__syncthreads();

			for (size_t j = 0; j < TILE; j++)
			{
				if (i + j < num)
				{
					acc = calculateAcceleration(entity, share_pos[j], acc);
				}
			}

			__syncthreads();
		}

		pos[idx] = updateLocation(entity, time, acc, speed);
		d_accs[idx] = acc;
		d_speed[idx] = updateSpeed(speed, acc, time);
	}
}

__global__
void bin_kernel(float3 *pos, float3 *accs, float4 *bin, int3 *offsets, size_t offset_len)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float3 entity = pos[idx];
	float3 acc = accs[idx];
	int x, y, z;

	x = (entity.x + OFFSET) / (POS_MAX - POS_MIN) * BIN_DIVISION;
	x = (x >= BIN_DIVISION) ? BIN_DIVISION - 1 : x;
	y = (entity.y + OFFSET) / (POS_MAX - POS_MIN) * BIN_DIVISION;
	y = (y >= BIN_DIVISION) ? BIN_DIVISION - 1 : y;
	z = (entity.z + OFFSET) / (POS_MAX - POS_MIN) * BIN_DIVISION;
	z = (z >= BIN_DIVISION) ? BIN_DIVISION - 1 : z;
	
	// go through offset list
	for (size_t i = 0; i < offset_len; i++)
	{
		int x_off = offsets[i].x;
		int y_off = offsets[i].y;
		int z_off = offsets[i].z;
		if (x+x_off < 0 || x+x_off >= BIN_DIVISION || y + y_off < 0 || y + y_off >= BIN_DIVISION || z + z_off < 0 || z + z_off >= BIN_DIVISION)
		{
			continue;
		}

		int index = to1D(x+x_off, y+y_off, z+z_off, BIN_DIVISION, BIN_DIVISION, BIN_DIVISION);

		// go through bin
		for (size_t j = 0; j < BIN_SIZE; j++)
		{
			float4 other = bin[index + j];
			if (other.w == 0)
			{
				break;
			}
			acc = calculateAcceleration(entity, make_float3(other.x, other.y, other.z), acc);
		}
	}
	accs[idx] = acc;
}

template<typename T>
void print(T* arr, int num)
{
	std::cout << "===== Start printing =====" << std::endl;
	for (size_t i = 0; i < num; i++)
	{
		std::cout << arr[i].x << ", " << arr[i].y << ", " << arr[i].z << std::endl;
	}
	std::cout << "===== End printing =====" << std::endl;
}


float3* cuda_caller(const glm::vec3 locations[], const glm::vec3 accs[])
{
	size_t bytes = sizeof(float3);
	float3 *d_locations, *d_accs, *h_locations, *h_accs;

	h_locations = (float3 *)malloc(bytes * NUM_OF_ENTITY);
	h_accs = (float3 *)malloc(bytes * NUM_OF_ENTITY);

	for (size_t i = 0; i < NUM_OF_ENTITY; i++)
	{
		h_locations[i] = make_float3(locations[i].x, locations[i].y, locations[i].z);
		h_accs[i] = make_float3(accs[i].x, accs[i].y, accs[i].z);
	}
	

	hipMalloc(&d_locations, bytes * NUM_OF_ENTITY);
	hipMalloc(&d_accs, bytes * NUM_OF_ENTITY);
	hipMemcpy(d_locations, h_locations, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);
	hipMemcpy(d_accs, h_accs, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);

	dim3 dimBlock(TILE);
	dim3 dimGrid(ceil((double)NUM_OF_ENTITY / (double)TILE));
	
	naive_kernel <<< dimGrid, dimBlock >>> (d_locations, d_accs, NUM_OF_ENTITY);

	hipMemcpy(h_accs, d_accs, bytes * NUM_OF_ENTITY, hipMemcpyDeviceToHost);
	hipFree(d_locations);
	hipFree(d_accs);
	free(h_locations);

	return h_accs;
}


float3* cuda_bin_caller(glm::vec4 bin[][BIN_SIZE], glm::vec3 locations[], glm::vec3 accs[], int3 *offset, size_t offset_len)
{
	size_t bytes = sizeof(float3);
	float3 *d_locations, *d_accs, *h_locations, *h_accs;
	float4 *d_bin, *h_bin;
	int3 *d_offsets;

	h_locations = (float3 *)malloc(bytes * NUM_OF_ENTITY);
	h_accs = (float3 *)malloc(bytes * NUM_OF_ENTITY);
	h_bin = (float4 *)malloc(sizeof(float4) * BIN_DIVISION * BIN_DIVISION * BIN_DIVISION * BIN_SIZE);
	

	for (size_t i = 0; i < NUM_OF_ENTITY; i++)
	{
		h_locations[i] = make_float3(locations[i].x, locations[i].y, locations[i].z);
		h_accs[i] = make_float3(accs[i].x, accs[i].y, accs[i].z);
	}

	for (size_t i = 0; i < BIN_DIVISION; i++)
	{
		for (size_t j = 0; j < BIN_DIVISION; j++)
		{
			for (size_t k = 0; k < BIN_DIVISION; k++)
			{
				int idx = to1D(i, j, k, BIN_DIVISION, BIN_DIVISION, BIN_DIVISION);
				for (size_t n = 0; n < BIN_SIZE; n++)
				{
					glm::vec4 entity = bin[idx][n];
					h_bin[idx + n] = make_float4(entity.x, entity.y, entity.z, entity.w);
				}
			}
		}
	}

	hipMalloc(&d_locations, bytes * NUM_OF_ENTITY);
	hipMalloc(&d_accs, bytes * NUM_OF_ENTITY);
	hipMalloc(&d_bin, BIN_DIVISION * BIN_DIVISION * BIN_DIVISION * BIN_SIZE * sizeof(float4));
	hipMalloc(&d_offsets, offset_len * sizeof(int3));

	hipMemcpy(d_locations, h_locations, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);
	hipMemcpy(d_accs, h_accs, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);
	hipMemcpy(d_bin, h_bin, BIN_DIVISION*BIN_DIVISION*BIN_DIVISION*BIN_SIZE * sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(d_offsets, offset, offset_len * sizeof(int3), hipMemcpyHostToDevice);

	dim3 dimBlock(TILE);
	dim3 dimGrid(ceil((double)NUM_OF_ENTITY / (double)TILE));

	bin_kernel <<< dimGrid, dimBlock >>> (d_locations, d_accs, d_bin, d_offsets, offset_len);

	hipMemcpy(h_accs, d_accs, bytes * NUM_OF_ENTITY, hipMemcpyDeviceToHost);
	hipFree(d_locations);
	hipFree(d_accs);
	hipFree(d_bin);
	hipFree(d_offsets);
	free(h_locations);
	free(h_bin);
	
	return h_accs;
}

void interop_caller(hipGraphicsResource **vbo_resource, float3 *d_speed, float3 *d_accs, float time)
{
	float3 *dptr;
	size_t bytes;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &bytes, *vbo_resource));

	dim3 dimBlock(TILE);
	dim3 dimGrid(ceil((double)NUM_OF_ENTITY / (double)TILE));

	naive_op_kernel <<< dimGrid, dimBlock >>> (dptr, d_speed, d_accs, time, NUM_OF_ENTITY);
	
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

