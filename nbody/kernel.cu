#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ 
float3 calculateAcceleration(float3 loc1, float3 loc2, float3 acc)
{
	float dx = loc2.x - loc1.x;
	float dy = loc2.y - loc1.y;
	float dz = loc2.z - loc1.z;
	float distSqrt = dx * dx + dy * dy + dz * dz + EPS2;
	float distCube = distSqrt * distSqrt * distSqrt;
	float invDist = G / sqrtf(distCube);
	acc.x += dx * invDist;
	acc.y += dy * invDist;
	acc.z += dz * invDist;
	acc.x *= SOFTEN;
	acc.y *= SOFTEN;
	acc.z *= SOFTEN;
	return acc;
}

__device__
float3 updateLocation(float3 loc, float time, float3 acc, float3 &speed)
{
	loc.x += speed.x * time + acc.x * (time * time / 2);
	loc.y += speed.y * time + acc.y * (time * time / 2);
	loc.z += speed.z * time + acc.z * (time * time / 2);

	if (loc.x < 0 - WALL || loc.x > WALL)
	{
		speed.x *= -1.0;
	}
	if (loc.y < 0 - WALL || loc.y > WALL)
	{
		speed.y *= -1.0;
	}
	if (loc.z < 0 - WALL || loc.z > WALL)
	{
		speed.z *= -1.0;
	}

	return loc;
}

__device__
float3 updateSpeed(float3 speed, float3 acc, float time)
{
	speed.x += acc.x * time;
	speed.y += acc.y * time;
	speed.z += acc.z * time;
	return speed;
}


__global__ 
void share_kernel(float3 *entities, float3 *accs, int length)
{
	unsigned int tx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + tx;
	__shared__ float3 share[TILE];
	float3 acc;
	float3 entity;

	if (idx < length)
	{
		entity = entities[idx];
		acc = accs[idx];
	
		for (size_t i = 0; i < length; i += TILE)
		{
			if (i + tx < length) 
			{
				share[tx] = entities[i + tx];
			}
			 
			__syncthreads();

			for (size_t j = 0; j < blockDim.x; j++)
			{
				if (i + j < length)
				{
					acc = calculateAcceleration(entity, share[j], acc);
				}
			}
			__syncthreads();
		}

		accs[idx] = acc;
	}
}

__global__ 
void naive_kernel(float3 *entities, float3 *accs, int length)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float3 entity, acc;
	if (idx < length)
	{
		entity = entities[idx];
		acc = accs[idx];


		for (size_t i = 0; i < length; i++)
		{
			acc = calculateAcceleration(entity, entities[i], acc);
		}
		accs[idx] = acc;
	}
}

__global__
void naive_kernel(float3 *pos, float3 *d_speed, float3 *d_accs, float time, int num)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num)
	{
		float3 entity = pos[idx];
		float3 speed = d_speed[idx];
		float3 acc = d_accs[idx];

		for (size_t i = 0; i < num; i++)
		{
			acc = calculateAcceleration(entity, pos[i], acc);
		}
		pos[idx] = updateLocation(entity, time, acc, speed);
		d_accs[idx] = acc;
		d_speed[idx] = updateSpeed(speed, acc, time);
	}
}

__global__
void share_kernel(float3 *pos, float3 *d_speed, float3 *d_accs, float time, int num)
{
	unsigned int thx = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + thx;
	float3 entity = pos[idx];
	float3 speed = d_speed[idx];
	float3 acc = d_accs[idx];
	__shared__ float3 share_pos[TILE];

	for (size_t i = 0; i < num; i += TILE)
	{
		share_pos[thx] = pos[i + thx];
		__syncthreads();
		for (size_t j = 0; j < TILE; j++)
		{
			acc = calculateAcceleration(entity, share_pos[j], acc);
		}
		__syncthreads();
	}

	d_accs[idx] = acc;
	d_speed[idx] = updateSpeed(speed, acc, time);
	pos[idx] = updateLocation(entity, time, acc, speed);
}

template<typename T>
void print(T* arr, int num)
{
	std::cout << "===== Start printing =====" << std::endl;
	for (size_t i = 0; i < num; i++)
	{
		std::cout << arr[i].x << ", " << arr[i].y << ", " << arr[i].z << std::endl;
	}
	std::cout << "===== End printing =====" << std::endl;
}


float3* cuda_caller(const glm::vec3 locations[], const glm::vec3 accs[])
{
	size_t bytes = sizeof(float3);
	float3 *d_locations, *d_accs, *h_locations, *h_accs;

	h_locations = (float3 *)malloc(bytes * NUM_OF_ENTITY);
	h_accs = (float3 *)malloc(bytes * NUM_OF_ENTITY);

	for (size_t i = 0; i < NUM_OF_ENTITY; i++)
	{
		h_locations[i] = make_float3(locations[i].x, locations[i].y, locations[i].z);
		h_accs[i] = make_float3(accs[i].x, accs[i].y, accs[i].z);
	}
	

	hipMalloc(&d_locations, bytes * NUM_OF_ENTITY);
	hipMalloc(&d_accs, bytes * NUM_OF_ENTITY);
	hipMemcpy(d_locations, h_locations, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);
	hipMemcpy(d_accs, h_accs, bytes * NUM_OF_ENTITY, hipMemcpyHostToDevice);

	dim3 dimBlock(TILE);
	dim3 dimGrid(ceil((double)NUM_OF_ENTITY / (double)TILE));
	
	share_kernel <<< dimGrid, dimBlock >>> (d_locations, d_accs, NUM_OF_ENTITY);

	hipMemcpy(h_accs, d_accs, bytes * NUM_OF_ENTITY, hipMemcpyDeviceToHost);
	hipFree(d_locations);
	hipFree(d_accs);
	free(h_locations);

	return h_accs;
}


void interop_caller(hipGraphicsResource **vbo_resource, float3 *d_speed, float3 *d_accs, float time)
{
	float3 *dptr;
	size_t bytes;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &bytes, *vbo_resource));

	dim3 dimBlock(TILE);
	dim3 dimGrid(ceil((double)NUM_OF_ENTITY / (double)TILE));

	share_kernel <<< dimGrid, dimBlock >>> (dptr, d_speed, d_accs, time, NUM_OF_ENTITY);
	
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}